#include "hip/hip_runtime.h"
#pragma once

#include "cuda_code.cuh"

#include <stdio.h>
#include<stdio.h>
#include<stdlib.h>
#include <math.h>
#include <Windows.h>
#include <time.h>
#include <assert.h>

#if USE_CPU_TIMER == 1
__int64 start, freq, end;
#define CHECK_TIME_START { QueryPerformanceFrequency((LARGE_INTEGER*)&freq); QueryPerformanceCounter((LARGE_INTEGER*)&start); }
#define CHECK_TIME_END(a) { QueryPerformanceCounter((LARGE_INTEGER*)&end); a = (float)((float)(end - start) / (freq / 1000.0f)); }
#else
#define CHECK_TIME_START
#define CHECK_TIME_END(a)
#endif

#if USE_GPU_TIMER == 1
hipEvent_t cuda_timer_start, cuda_timer_stop;
#define CUDA_STREAM_0 (0)

void create_device_timer()
{
	CUDA_CALL(hipEventCreate(&cuda_timer_start));
	CUDA_CALL(hipEventCreate(&cuda_timer_stop));
}

void destroy_device_timer()
{
	CUDA_CALL(hipEventDestroy(cuda_timer_start));
	CUDA_CALL(hipEventDestroy(cuda_timer_stop));
}

inline void start_device_timer()
{
	hipEventRecord(cuda_timer_start, CUDA_STREAM_0);
}

inline TIMER_T stop_device_timer()
{
	TIMER_T ms;
	hipEventRecord(cuda_timer_stop, CUDA_STREAM_0);
	hipEventSynchronize(cuda_timer_stop);

	hipEventElapsedTime(&ms, cuda_timer_start, cuda_timer_stop);
	return ms;
}

#define CHECK_TIME_INIT_GPU() { create_device_timer(); }
#define CHECK_TIME_START_GPU() { start_device_timer(); }
#define CHECK_TIME_END_GPU(a) { a = stop_device_timer(); }
#define CHECK_TIME_DEST_GPU() { destroy_device_timer(); }
#else
#define CHECK_TIME_INIT_GPU()
#define CHECK_TIME_START_GPU()
#define CHECK_TIME_END_GPU(a)
#define CHECK_TIME_DEST_GPU()
#endif

TIMER_T compute_time = 0;
TIMER_T device_time = 0;

#define Window 2
#define BLOCK_SIZE (1<<5)
#define FILTER_SIZE 5

__constant__ float constant_gaussian_kernel[ 25 ];

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//	
//	Gaussian ���͸��� �ϴ� Ŀ��
//	shared memory�� ������� �ʴ´�
//	
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void Gaussian_kernel_no_shared(IN unsigned char *d_bitmaps, OUT unsigned char *d_Gaussian, long width, long height) {	
	const unsigned block_id = blockIdx.y*gridDim.x + blockIdx.x;
	const unsigned thread_id = threadIdx.y*blockDim.x + threadIdx.x;
	const unsigned id = block_id * BLOCK_SIZE + thread_id;

	double sum = 0;
	int index1, index2;
	int row = FILTER_SIZE, col = FILTER_SIZE;

	for (int i = 0; i < width; i++) {
		for (int j = 0; j < row; j++) {
			for (int k = 0; k < col; k++) {
				index1 = id + j - row / 2;
				index2 = i + k - col / 2;
				if (index1 < 0 || index1 >= height) continue;
				if (index2 < 0 || index2 >= width) continue;
				sum += d_bitmaps[index1*width + index2] * constant_gaussian_kernel[j*row + k];
			}
		}
		d_Gaussian[id*width + i] = sum;
		sum = 0;
	}
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//	
//	Gaussian ���͸��� �ϴ� Ŀ��
//	shared memory�� ����Ѵ�.
//	
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
extern __shared__ unsigned char sharedBuffer[];
__global__ void Gaussian_kernel_shared(INOUT unsigned char *d_bitmaps, OUT unsigned char *d_Gaussian, long width, long height) {
	const unsigned block_id = blockIdx.y*gridDim.x + blockIdx.x;
	const unsigned thread_id = threadIdx.y*blockDim.x + threadIdx.x;
	const unsigned id = block_id * BLOCK_SIZE + thread_id;

	double sum = 0;
	int index;
	int row = FILTER_SIZE, col = FILTER_SIZE;

	if (thread_id == 0) {
		for (int i = 0; i < width; i++) {
			for (int j = 0; j < row / 2; j++) {
				index = id - j - 1;
				if (index < 0) sharedBuffer[(row / 2 - j - 1)*width + i] = 0;
				else sharedBuffer[(row / 2 - j - 1)*width + i] = d_bitmaps[index*width + i];
			}
		}
	}
	if (thread_id == BLOCK_SIZE - 1) {
		for (int i = 0; i < width; i++) {
			for (int j = BLOCK_SIZE; j < BLOCK_SIZE + row / 2; j++) {
				index = id + j - BLOCK_SIZE + 1;
				if (index > height) sharedBuffer[(row / 2 + j)*width + i] = 0;
				else sharedBuffer[(row / 2 + j)*width + i] = d_bitmaps[index*width + i];
			}
		}
	}

	for (int i = 0; i < width; i++) {
		sharedBuffer[(thread_id + row / 2)*width + i] = d_bitmaps[id*width + i];
	}

	__syncthreads();

	for (int i = 0; i < width; i++) {
		for (int j = 0; j < row; j++) {
			for (int k = 0; k < col; k++) {
				index = i + k - col / 2;
				if (index < 0 || index >= width) continue;
				sum += sharedBuffer[(thread_id + j)*width + index] * constant_gaussian_kernel[j*row + k + row/2];
			}
		}
		d_Gaussian[id*width + i] = sum;
		sum = 0;
	}
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//	
//	Constant variable �� gaussian kernel�� �����ϴ� �Լ�
//	�Ŀ� gaussian filtering ���� ����Ѵ�.
//	
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

void Set_Gaussain_Kernel(){
	float _1 = 1.0f / 256.0f;
	float _4 = _1 * 4;
	float _6 = _1 * 6;
	float _16 = _1 * 16;
	float _24 = _1 * 24;
	float _36 = _1 * 36;

	float *p_gaussian_kernel = new float[25];

	p_gaussian_kernel[0] = p_gaussian_kernel[4] = p_gaussian_kernel[20] = p_gaussian_kernel[24] = _1;
	p_gaussian_kernel[1] = p_gaussian_kernel[3] = p_gaussian_kernel[5] = p_gaussian_kernel[9]= _4;
	p_gaussian_kernel[15] = p_gaussian_kernel[19] = p_gaussian_kernel[21] = p_gaussian_kernel[23] = _4;
	p_gaussian_kernel[2] = p_gaussian_kernel[10] = p_gaussian_kernel[14] = p_gaussian_kernel[22] = _6;
	p_gaussian_kernel[6] = p_gaussian_kernel[8] = p_gaussian_kernel[16] = p_gaussian_kernel[18] = _16;
	p_gaussian_kernel[7] = p_gaussian_kernel[11] =p_gaussian_kernel[13] = p_gaussian_kernel[17] = _24;
	p_gaussian_kernel[12] = _36;

	hipMemcpyToSymbol(HIP_SYMBOL( constant_gaussian_kernel), p_gaussian_kernel, sizeof( float ) * 25 );

	delete[] p_gaussian_kernel;
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//	
//	Ŀ���� �����ϱ� �� �ʿ��� �ڷ�� �غ� �� Ŀ���� ������ ����̽��� ����
//	Shared_flag �Է� �� NO_SHARED �� SHARED �� �� ���� ��ũ�θ� ������
//	flag���� �´� Ŀ���� ����
//	
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

float Do_Gaussian_on_GPU(IN unsigned char *p_bitmaps, OUT unsigned char *p_Gaussian, long width, long height, int Shared_flag)
{
	Set_Gaussain_Kernel();
	CUDA_CALL(hipSetDevice(0));
	unsigned int total_pixel = width * height;

	unsigned char *d_bitmaps, *d_Gaussian;
	size_t mem_size;

	mem_size = width * height * sizeof(unsigned char);
	CUDA_CALL(hipMalloc(&d_bitmaps, mem_size));

	CUDA_CALL(hipMemcpy(d_bitmaps, p_bitmaps, mem_size, hipMemcpyHostToDevice));

	CUDA_CALL(hipMalloc(&d_Gaussian, mem_size));

	dim3 blockDim(32, 32);

	dim3 gridDim((width + 31) / blockDim.x, (height + 31) / blockDim.y); 

	dim3 dimBlock(BLOCK_SIZE, 1);
	dim3 dimGrid(height / dimBlock.x, 1);

	CHECK_TIME_INIT_GPU();
	CHECK_TIME_START_GPU();

	switch (Shared_flag)
	{
	case NO_SHARED:
		Gaussian_kernel_no_shared << <gridDim, blockDim >> > (d_bitmaps, d_Gaussian, width, height);
		break;
	case SHARED:
		Gaussian_kernel_shared << < dimGrid, dimBlock, sizeof(unsigned char) * (dimBlock.x + 2 * Window) * width >> > (d_bitmaps, d_Gaussian, width, height);
		break;
	}

	CUDA_CALL(hipDeviceSynchronize());
	CHECK_TIME_END_GPU(device_time);
	CHECK_TIME_DEST_GPU();

	CUDA_CALL(hipMemcpy(p_Gaussian, d_Gaussian, mem_size, hipMemcpyDeviceToHost));
	
	hipFree(d_bitmaps);
	hipFree(d_Gaussian);

	return device_time;
}