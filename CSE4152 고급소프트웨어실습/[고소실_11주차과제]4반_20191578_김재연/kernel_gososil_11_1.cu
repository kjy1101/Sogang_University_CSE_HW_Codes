#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include<stdio.h>
#include<stdlib.h>
#include <math.h>
#include <Windows.h>
#include <time.h>
#include <assert.h>

#define CUDA_CALL(x) { const hipError_t a = (x); if(a != hipSuccess) { printf("\nCuda Error: %s (err_num=%d) at line:%d\n", hipGetErrorString(a), a, __LINE__); hipDeviceReset(); assert(0);}}
typedef float TIMER_T;

#define USE_CPU_TIMER 1
#define USE_GPU_TIMER 1

#if USE_CPU_TIMER == 1
__int64 start, freq, end;
#define CHECK_TIME_START { QueryPerformanceFrequency((LARGE_INTEGER*)&freq); QueryPerformanceCounter((LARGE_INTEGER*)&start); }
#define CHECK_TIME_END(a) { QueryPerformanceCounter((LARGE_INTEGER*)&end); a = (float)((float)(end - start) / (freq / 1000.0f)); }
#else
#define CHECK_TIME_START
#define CHECK_TIME_END(a)
#endif

#if USE_GPU_TIMER == 1
hipEvent_t cuda_timer_start, cuda_timer_stop;
#define CUDA_STREAM_0 (0)

void create_device_timer()
{
    CUDA_CALL(hipEventCreate(&cuda_timer_start));
    CUDA_CALL(hipEventCreate(&cuda_timer_stop));
}

void destroy_device_timer()
{
    CUDA_CALL(hipEventDestroy(cuda_timer_start));
    CUDA_CALL(hipEventDestroy(cuda_timer_stop));
}

inline void start_device_timer()
{
    hipEventRecord(cuda_timer_start, CUDA_STREAM_0);
}

inline TIMER_T stop_device_timer()
{
    TIMER_T ms;
    hipEventRecord(cuda_timer_stop, CUDA_STREAM_0);
    hipEventSynchronize(cuda_timer_stop);

    hipEventElapsedTime(&ms, cuda_timer_start, cuda_timer_stop);
    return ms;
}

#define CHECK_TIME_INIT_GPU() { create_device_timer(); }
#define CHECK_TIME_START_GPU() { start_device_timer(); }
#define CHECK_TIME_END_GPU(a) { a = stop_device_timer(); }
#define CHECK_TIME_DEST_GPU() { destroy_device_timer(); }
#else
#define CHECK_TIME_INIT_GPU()
#define CHECK_TIME_START_GPU()
#define CHECK_TIME_END_GPU(a)
#define CHECK_TIME_DEST_GPU()
#endif

#define BLOCK_SIZE 32
#define DATA_SIZE (1 << 26)

TIMER_T compute_time = 0;
TIMER_T device_time = 0;

int N;
int N_EQUATIONS;
float(*A); float(*B); float(*C);
float(*X0); float(*X1); float(*FX0); float(*FX1);
float(*X0_GPU); float(*X1_GPU); float(*FX0_GPU); float(*FX1_GPU);

hipError_t find_roots_GPU(float *A, float *B, float *C, float *X0_GPU, float *X1_GPU, float *FX0_GPU, float *FX1_GPU);

__global__ void find_roots_Kernel(float* d_A, float* d_B, float* d_C, float* d_X0, float* d_X1, float* d_FX0, float* d_FX1)
{
    int i;
    float a, b, c, d, x0, x1, tmp;
    i = blockDim.x * blockIdx.x + threadIdx.x;
    a = d_A[i]; b = d_B[i]; c = d_C[i];
    d = sqrtf(b * b - 4.0f * a * c);
    tmp = 1.0f / (2.0f * a);
    d_X0[i] = x0 = (-b - d) * tmp;
    d_X1[i] = x1 = (-b + d) * tmp;
    d_FX0[i] = (a * x0 + b) * x0 + c;
    d_FX1[i] = (a * x1 + b) * x1 + c;
}

void find_roots_CPU(float* A, float* B, float* C, float* X0, float* X1, float* FX0, float* FX1, int n) {
    int i;
    float a, b, c, d, x0, x1, tmp;
    for (i = 0; i < n; i++) {
        a = A[i]; b = B[i]; c = C[i];
        d = sqrtf(b * b - 4.0f * a * c);
        tmp = 1.0f / (2.0f * a);
        X0[i] = x0 = (-b - d) * tmp;
        X1[i] = x1 = (-b + d) * tmp;
        FX0[i] = (a * x0 + b) * x0 + c;
        FX1[i] = (a * x1 + b) * x1 + c;
    }
}

void init_bin_file(int size) {
    srand((unsigned)time(NULL));
    printf("***Binary File init Start!!\n");
    FILE* fpA = fopen("A.bin", "wb");
    FILE* fpB = fopen("B.bin", "wb");
    FILE* fpC = fopen("C.bin", "wb");

    fwrite(&size, sizeof(int), 1, fpA);
    fwrite(&size, sizeof(int), 1, fpB);
    fwrite(&size, sizeof(int), 1, fpC);

    float random_num;
    for (int i = 0; i < size; i++) {
        random_num = (float)(((double)rand() / RAND_MAX) * 30.0f);
        fwrite(&random_num, sizeof(float), 1, fpA);
        random_num = (float)(((double)rand() / RAND_MAX) * 40.0f);
        fwrite(&random_num, sizeof(float), 1, fpB);
        random_num = (float)(((double)rand() / RAND_MAX) * 50.0f);
        fwrite(&random_num, sizeof(float), 1, fpC);
    }

    fclose(fpA);
    fclose(fpB);
    fclose(fpC);
    printf("***Binary File init End!!\n\n");
}

void read_bin_file()
{
    printf("***Binary File Read Start!!\n");
    FILE* fpA = fopen("A.bin", "rb");
    FILE* fpB = fopen("B.bin", "rb");
    FILE* fpC = fopen("C.bin", "rb");
    
	fread(&N_EQUATIONS, sizeof(int), 1, fpA);
	fread(&N_EQUATIONS, sizeof(int), 1, fpB);
	fread(&N_EQUATIONS, sizeof(int), 1, fpC);

    A = new float[N_EQUATIONS];
    B = new float[N_EQUATIONS];
    C = new float[N_EQUATIONS];
    X0 = new float[N_EQUATIONS];
    X1 = new float[N_EQUATIONS];
    FX0 = new float[N_EQUATIONS];
    FX1 = new float[N_EQUATIONS];
    X0_GPU = new float[N_EQUATIONS];
    X1_GPU = new float[N_EQUATIONS];
    FX0_GPU = new float[N_EQUATIONS];
    FX1_GPU = new float[N_EQUATIONS];

    fread(A, sizeof(float), N_EQUATIONS, fpA);
    fread(B, sizeof(float), N_EQUATIONS, fpB);
    fread(C, sizeof(float), N_EQUATIONS, fpC);

    fclose(fpA);
    fclose(fpB);
    fclose(fpC);
    printf("***Binary File Read End!!\n\n");
}

int main()
{
    init_bin_file(DATA_SIZE);
    read_bin_file();

    printf("N_EQUATIONS: %d, Block size: %d\n", N_EQUATIONS, BLOCK_SIZE);
    int i;

    //CPU 
    printf("***find_roots_CPU Start!!\n");
    CHECK_TIME_START;
    find_roots_CPU(A, B, C, X0, X1, FX0, FX1, N_EQUATIONS);
    CHECK_TIME_END(compute_time);
    printf("***find_roots_CPU End!!\n\n");

    printf("CPU time = %.6f\n", compute_time);
    //GPU

    printf("***find_roots_GPU Start!!\n");
    find_roots_GPU(A, B, C, X0_GPU, X1_GPU, FX0_GPU, FX1_GPU);
    printf("***find_roots_GPU End!!\n\n");

    printf("GPU time = %.6f\n", device_time);

    /*for (i = 0; i < N_EQUATIONS; i++)
    {
        if (FX0[i] != FX0_GPU[i])
        {
            printf("[0] %u != %u\n", FX0[i], FX0_GPU[i]);
            break;
        }
        else if (FX1[i] != FX1_GPU[i]) 
        {
            printf("[1] %u != %u\n", FX1[i], FX1_GPU[i]);
            break;
        }
    }*/

    if (i == N_EQUATIONS)
        printf("***Kernel execution Success!!\n\n");

    // Write the output array into the output file.
    FILE *fpX0 = fopen("X0.bin", "wb");
    FILE* fpX1 = fopen("X1.bin", "wb");
    FILE* fpFX0 = fopen("FX0.bin", "wb");
    FILE* fpFX1 = fopen("FX1.bin", "wb");
    if (!fpX0 || !fpX1 || !fpFX0 || !fpFX1 ) {
        fprintf(stderr, "Error: cannot open the output file...\n");
        exit(-1);
    }

    N = N_EQUATIONS;
    fwrite(&N, sizeof(int), 1, fpX0);
    fwrite(&N, sizeof(int), 1, fpX1);
    fwrite(&N, sizeof(int), 1, fpFX0);
    fwrite(&N, sizeof(int), 1, fpFX1);

    for (i = 0; i < N_EQUATIONS; i++) {
        fwrite(&X0_GPU[i], sizeof(float), 1, fpX0);
        fwrite(&X1_GPU[i], sizeof(float), 1, fpX1);
        fwrite(&FX0_GPU[i], sizeof(float), 1, fpFX0);
        fwrite(&FX1_GPU[i], sizeof(float), 1, fpFX1);
    }


    fclose(fpX0);
    fclose(fpX1);
    fclose(fpFX0);
    fclose(fpFX1);
    printf("end!!\n\n");

    return 0;
}


hipError_t find_roots_GPU(float* A, float* B, float* C, float* X0_GPU, float* X1_GPU, float* FX0_GPU, float* FX1_GPU)
{
    CHECK_TIME_INIT_GPU();

    hipError_t cudaStatus;
    
    float(*p_A); float(*p_B); float(*p_C);
    float(*p_X0); float(*p_X1); float(*p_FX0); float(*p_FX1);
    size_t size;

    size = N_EQUATIONS * sizeof(float);
    CUDA_CALL(hipMalloc(&p_A, size))
    CUDA_CALL(hipMemcpy(p_A, A, size, hipMemcpyHostToDevice))
    CUDA_CALL(hipMalloc(&p_B, size))
    CUDA_CALL(hipMemcpy(p_B, B, size, hipMemcpyHostToDevice))
    CUDA_CALL(hipMalloc(&p_C, size))
    CUDA_CALL(hipMemcpy(p_C, C, size, hipMemcpyHostToDevice))

    CUDA_CALL(hipMalloc(&p_X0, size))
    CUDA_CALL(hipMalloc(&p_X1, size))
    CUDA_CALL(hipMalloc(&p_FX0, size))
    CUDA_CALL(hipMalloc(&p_FX1, size))

    dim3 dimBlock(BLOCK_SIZE, 1);
    dim3 dimGrid(N_EQUATIONS / dimBlock.x, 1);

    CHECK_TIME_START_GPU();
    find_roots_Kernel << < dimGrid, dimBlock >> > (p_A, p_B, p_C, p_X0, p_X1, p_FX0, p_FX1);
    CHECK_TIME_END_GPU(device_time);

    CUDA_CALL(hipDeviceSynchronize())
        CUDA_CALL(hipMemcpy(X0_GPU, p_X0, size, hipMemcpyDeviceToHost))
        CUDA_CALL(hipMemcpy(X1_GPU, p_X1, size, hipMemcpyDeviceToHost))
        CUDA_CALL(hipMemcpy(FX0_GPU, p_FX0, size, hipMemcpyDeviceToHost))
        CUDA_CALL(hipMemcpy(FX1_GPU, p_FX1, size, hipMemcpyDeviceToHost))

    hipFree(p_A);
    hipFree(p_B);
    hipFree(p_C);
    hipFree(p_X0);
    hipFree(p_X1);
    hipFree(p_FX0);
    hipFree(p_FX1);
    CHECK_TIME_DEST_GPU();
    return cudaStatus;
}